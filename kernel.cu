#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>

// Include the CUDA runtime API
#include <hip/hip_runtime.h>

// Define the vector size
#define N 100000000

// Define the CPU function for adding two vectors
void add_cpu(long long* a, long long* b, long long* c) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

// Define the CUDA kernel for adding two vectors
__global__ void add_gpu(long long* a, long long* b, long long* c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    // Allocate memory for the vectors using hipMallocManaged
    long long* a, * b, * c_cpu, * c_gpu;
    hipMallocManaged(&a, N * sizeof(long long));
    hipMallocManaged(&b, N * sizeof(long long));
    hipMallocManaged(&c_cpu, N * sizeof(long long));
    hipMallocManaged(&c_gpu, N * sizeof(long long));

    // Initialize the vectors with some values
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
        c_cpu[i] = 0;
        c_gpu[i] = 0;
    }

    // Measure the elapsed time for the CPU implementation
    auto start_time_cpu = std::chrono::high_resolution_clock::now();
    add_cpu(a, b, c_cpu);
    auto end_time_cpu = std::chrono::high_resolution_clock::now();
    auto elapsed_time_cpu = std::chrono::duration_cast<std::chrono::microseconds>(end_time_cpu - start_time_cpu).count();

    // Measure the elapsed time for the CUDA implementation
    auto start_time_gpu = std::chrono::high_resolution_clock::now();
    add_gpu << <1, N >> > (a, b, c_gpu);
    hipDeviceSynchronize();
    auto end_time_gpu = std::chrono::high_resolution_clock::now();
    auto elapsed_time_gpu = std::chrono::duration_cast<std::chrono::microseconds>(end_time_gpu - start_time_gpu).count();

    // Check that the CPU and GPU results are the same
    for (int i = 0; i < N; i++) {
        if (c_cpu[i] != c_gpu[i]) {
            std::cerr << "Error: CPU and GPU results differ at index " << i << std::endl;
            break;
        }
    }

    // Print the elapsed time for the CPU and GPU implementations
    std::cout << "Elapsed time (CPU): " << elapsed_time_cpu << " microseconds" << std::endl;
    std::cout << "Elapsed time (GPU): " << elapsed_time_gpu << " microseconds" << std::endl;

    // Free the memory for the vectors using hipFree
    hipFree(a);
    hipFree(b);
    hipFree(c_cpu);
    hipFree(c_gpu);

    return 0;
}
